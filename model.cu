#include "hip/hip_runtime.h"
/*
  Copyright (c) 2011 Paul Richmond, University of Sheffield , UK; 
  all rights reserved unless otherwise stated.

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  In addition to the regulations of the GNU General Public License,
  publications and communications based in parts on this program or on
  parts of this program are required to cite the article 
  "Democratic population decisions result in robust policy-gradient 
  learning: a parametric study with GPU simulations" by Paul Richmond, 
  Lars Buesing, Michele Giugliano and Eleni Vasilaki, PLoS ONE Neuroscience, 
  Under Review.. 

  This program is distributed in the hope that it will be useful, but
  WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
  General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston,
  MA 02111-1307 USA
*/


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <windows.h>
#include <time.h>
#include <float.h>
#include <cutil_inline.h>

// includes, project
#include "reduction.cuh"
#include "model.h"
#include "model.cuh"
#include "output.cuh"

////////////////////////////////////////////////////////////////////////////////
//Macro definitions
#define MAX(x,y) ((x > y) ? x : y)
#define MAX_NEURONS MAX(M, N)
#define MAX_RAND MAX_NEURONS //MAX(MAX_NEURONS, SM_CHUNK_SIZE*SM_CHUNK_SIZE)	//factor of T required for no dyn, N*M for weight update noise



bool printWeightPlot;

////////////////////////////////////////////////////////////////////////////////
// Thread/Grid blocks
dim3 m_threads;	
dim3 m_grid;

dim3 n_threads;	
dim3 n_grid;

dim3 nN_threads;
dim3 nN_grid;

dim3 mN_threads;
dim3 mN_grid;

dim3 NT_matrix_threads;	
dim3 NT_matrix_grid;	
int NT_grid_width;

dim3 NT_threads;
dim3 NT_grid;

dim3 MN_matrix_threads;
dim3 MN_matrix_grid;
int MN_grid_width;

dim3 MN_threads;
dim3 MN_grid;

//////////////////////////////////////////////////////
//Persitant global variables
float* d_Wr;
float* d_W;
float* d_W_out;
float* d_Wt;
//Step variables
float* h_In_Deg;	//host data
float* d_In_Deg;
float* d_rew;
int* d_decision_offset;
int* d_target_offset;
//Trial variables
float* d_u;
float* d_u_out;
float* d_Y;
float* d_Yt;
float* d_Y_sum;
float* d_YProbt;
float* d_input_pot;
float* d_lateral_pot;
float* d_YProb;
float* d_X;	
float* d_Input;	
float* d_epsp;
float* d_Grad;
float* d_out_x;
float* d_out_x_sum;
float* d_out_y;
float* d_out_y_sum;
rand48seeds* d_randSeeds;
magic_numbers mn;

//data used for producing gradiant graphs
float* h_W;
float* d_sumGrad;
float* d_sumDeltaW;
float* d_sumGrad_out;
float* d_sumDeltaW_out;
float* h_sumGrad;
float* h_sumDeltaW;



/**	Reward function  */
float reward(float x, int config)
{
	float reward = 0;

	//guassian reward
	if (reward_func == GAUSSIAN)
	{
		if (x<PI)
			reward = exp(-pow(x, 2.0f)/(2.0f*pow(sigma_R[config], 2.0f)));
		else
			reward = exp(-pow(2.0f*PI-x, 2.0f)/(2.0f*pow(sigma_R[config], 2.0f))); 

	}
	//box reward
	else if (reward_func == BOX)
	{
		if (x<PI)
			reward = (float)(x<sigma_R_box[config])*(x>-sigma_R_box[config]);
		else
			reward = (float)((2.0f*PI-x)<sigma_R_box[config]) * ((2.0f*PI-x)>-sigma_R_box[config]);
	}

	return reward;
}


float errorFunc(float x)
{

	float err;
	
	if (x<PI)
		err = x;
	else
		err = (2.0f*PI-x);

	return err;
}

/* Mod function using sign from divisor (as with python)*/
float py_modf(float n, float a) {
    float r = fmodf(n, a);
	float sign = (a > 0) ? 1.0f : -1.0f;
    if (r * sign < 0) r += a;
    return r;
}


void copyParametersToDevice()
{
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(beta), &h_beta, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(xsc), &h_xsc, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(u0), &h_u0, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(w0), &h_w0, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(eta), &h_eta, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(baseline), &h_baseline, sizeof(float)*ind_configs));

	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(tau), &h_tau, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(lambda), &h_lambda, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(urest), &h_urest, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(threshold), &h_threshold, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(du), &h_du, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(rho0), &h_rho0, sizeof(float)*ind_configs));
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(ref), &h_ref, sizeof(float)*ind_configs));
}


/** initLearnStepData
 * Initialises all device data used during the learn_step function.
 * Initialises Weight matrices W (+W_out) and Wr on the device. Wr has static laternal connection weights. W is set to zero. 
 * A seperate set of data is created for each indipendant trial excluding Wr which is the same throughout all.
 * A set of host data is allocated (h_In_Deg) which is used in the learn step to create random goals.
 */
void initLearnStepData()
{
	//allocate W matrix on host to save weights later on
	if (printWeightPlot)
	{
		h_W = (float*) malloc(N*M*sizeof(float)*ind_trials);
	}

	//allocate weight data
	hipMalloc( (void**) &d_Wr, N*N*sizeof(float)*ind_configs );
	hipMalloc( (void**) &d_W, M*N*sizeof(float)*ind_trials);
	hipMalloc( (void**) &d_W_out, M*N*sizeof(float) *ind_trials);
	hipMalloc( (void**) &d_Wt, M*N*sizeof(float) *ind_trials);

	//init lateral connection weights and copy to device
	float* h_Wr = (float*) malloc(N*N*sizeof(float)*ind_configs);
	for(int i=0; i<N; i++)
	{
		for (int j=0; j<N; j++)
		{
			float dist = abs(j-i*1.0f); 
			dist = dist*(dist<(N/2)) + (N-dist)*(dist>=(N/2));

			//Wr varies for each set of configuration parameters
			for(int k=0; k<ind_configs; k++)
			{
				h_Wr[j+(i*N) + (k*N*N)] =	(expf(-powf(dist,2.0f)/(2.0f*powf(sig_p[k],2.0f)))*w_E[k]  - 0.9f)*rsc[k];		//j+(i*N) possibly should be i+(j*N)
			}
		}
	}
	cutilSafeCall( hipMemcpy(d_Wr, h_Wr, N*N*sizeof(float)*ind_configs, hipMemcpyHostToDevice) );
	free(h_Wr);

	//Set action cell weights to zero
	cutilSafeCall( hipMemset(d_W, 0,M*N*sizeof(float)*ind_trials));
	cutilSafeCall( hipMemset(d_W_out, 0,M*N*sizeof(float)*ind_trials));
	cutilSafeCall( hipMemset(d_Wt, 0,M*N*sizeof(float)*ind_trials));

	//allocate global learnstep variables
	hipMalloc( (void**) &d_In_Deg, Tr_MAX*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_rew, sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_decision_offset, sizeof(int)*ind_trials );
	hipMalloc( (void**) &d_target_offset, sizeof(int)*ind_trials );
	

	//allocate trial specific data
	hipMalloc( (void**) &d_u, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_u_out, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_Y, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_Yt, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_Y_sum, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_YProb, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_YProbt, T*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_input_pot, T*N*sizeof(float)*ind_trials );	//factor of T only required for no dyn system
	hipMalloc( (void**) &d_lateral_pot, N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_X, T*M*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_Input, T*M*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_epsp, T*M*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_Grad, M*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_out_x, N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_out_x_sum, N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_out_y, N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_out_y_sum, N*sizeof(float)*ind_trials );

	//init host data used to create a set of random goals 
	h_In_Deg = (float*) malloc(Tr_MAX*sizeof(float)*ind_trials);

	//init randomn number stream data using rand48 algorithm
	rand48seeds* h_randSeeds;
	h_randSeeds = (rand48seeds*) malloc(MAX_RAND*sizeof(rand48seeds)*ind_trials);
	hipMalloc( (void**) &d_randSeeds, MAX_RAND*sizeof(rand48seeds)*ind_trials);
	initCUDARand48(MAX_RAND*ind_trials, h_randSeeds, d_randSeeds, mn);
	free(h_randSeeds);
}

void initGraphAnalysis()
{
	//malloc data on host
	h_sumGrad = (float*)malloc(M*N*sizeof(float)*ind_trials);
	h_sumDeltaW = (float*)malloc(M*N*sizeof(float)*ind_trials);
	
	//malloc dat on device
	hipMalloc( (void**) &d_sumGrad, M*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_sumDeltaW, M*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_sumGrad_out, M*N*sizeof(float)*ind_trials );
	hipMalloc( (void**) &d_sumDeltaW_out, M*N*sizeof(float)*ind_trials );
}

void resetGraphAnalysis()
{
	//block set device data to 0
	cutilSafeCall( hipMemset(d_sumGrad, 0,M*N*sizeof(float)*ind_trials));
	cutilSafeCall( hipMemset(d_sumDeltaW, 0,M*N*sizeof(float)*ind_trials));
	cutilSafeCall( hipMemset(d_sumGrad_out, 0,M*N*sizeof(float)*ind_trials));
	cutilSafeCall( hipMemset(d_sumDeltaW_out, 0,M*N*sizeof(float)*ind_trials));
}

void cleanupGraphAnalysis()
{
	//free host memory
	free(h_sumGrad);
	free(h_sumDeltaW);

	//free device memory
	hipFree(d_sumGrad);
	hipFree(d_sumDeltaW);
	hipFree(d_sumGrad_out);
	hipFree(d_sumDeltaW_out);
}


/** cleanupLearnStep
 *	Frees all data on host and device which is used in the learn step.
 */
void cleanupLearnStep()
{
	//dealloc learn step data
	free(h_In_Deg); 
	hipFree(d_In_Deg);
	hipFree(d_rew);
	hipFree(d_decision_offset);
	hipFree(d_target_offset);
	

	//dealloc trial data
	hipFree (d_randSeeds);
	hipFree(d_u);
	hipFree(d_u_out);
	hipFree(d_Y);
	hipFree(d_Yt);
	hipFree(d_Y_sum);
	hipFree(d_YProb);
	hipFree(d_YProbt);
	hipFree(d_input_pot);
	hipFree(d_lateral_pot);
	hipFree(d_X);
	hipFree(d_Input);
	hipFree(d_epsp);
	hipFree(d_Grad);
	hipFree(d_out_x);
	hipFree(d_out_x_sum);
	hipFree(d_out_y);
	hipFree(d_out_y_sum);

	//dealloc weight data
	if (printWeightPlot)
		free(h_W);
	hipFree( d_W);
	hipFree( d_W_out);
	hipFree( d_Wr);
	hipFree( d_Wt);

}

/** calculateGridBlockSizes
 * All grid block sizes are pre-calculated before the learn step
 */
void calculateGridBlockSizes()
{
	//M total threads with a max block size of M
	m_threads = dim3(M_BLOCK_SIZE, 1, 1);	
	m_grid = dim3(M/M_BLOCK_SIZE, ind_trials, 1);

	//N total threads with a max block size of N
	n_threads = dim3(N_BLOCK_SIZE, 1, 1);	
	n_grid = dim3(N/N_BLOCK_SIZE, ind_trials, 1);

	//n block size by N total threads
	nN_threads = dim3(N_BLOCK_SIZE, 1 , 1);
	nN_grid = dim3(N, ind_trials, 1);

	//m block size by N total threads
	mN_threads = dim3(M_BLOCK_SIZE,1 , 1);
	mN_grid = dim3(N, ind_trials, 1);

	//NxT total threads with SM_CHUNK_SIZE^2 threads per block
	NT_matrix_threads = dim3(SM_CHUNK_SIZE, SM_CHUNK_SIZE, 1);	
	NT_matrix_grid = dim3(N*T/(SM_CHUNK_SIZE*SM_CHUNK_SIZE), ind_trials, 1);
	NT_grid_width = N/SM_CHUNK_SIZE;

	//N*T total threads
	NT_threads = dim3(SM_CHUNK_SIZE *SM_CHUNK_SIZE, 1, 1);	
	NT_grid = dim3(N*T/(SM_CHUNK_SIZE*SM_CHUNK_SIZE), ind_trials, 1);

	//NxM total threads with SM_CHUNK_SIZE^2 threads per 2D block
	//both x and y of grid are held within the x dimenion of the grid (requires mod and divide in kernel)
	MN_matrix_threads = dim3(SM_CHUNK_SIZE, SM_CHUNK_SIZE, 1);
	MN_matrix_grid = dim3(M*N/(SM_CHUNK_SIZE*SM_CHUNK_SIZE), ind_trials, 1);
	MN_grid_width = M/SM_CHUNK_SIZE;

	//NxM total threads with SM_CHUNK_SIZE^2 threads per 1D block
	MN_threads = dim3(SM_CHUNK_SIZE*SM_CHUNK_SIZE, 1, 1);
	MN_grid = dim3(M*N/(SM_CHUNK_SIZE*SM_CHUNK_SIZE), ind_trials, 1);
}

/**
 *	learn_step function
 */
template <LEARNING learning, DYNAMICS dynamics, PROFILING profiling>
void learn_step(int Tr, float* total_reward, float* total_error)
{
	

	//global learn step variables
	for (int i=0; i<ind_trials; i++)
	{
		total_reward[i] = 0;
		total_error[i] = 0;
	}

	srand ( (unsigned int)time(NULL) );
	for (int i=0;i<Tr*ind_trials;i++)
	{
		if (moving_target)
			h_In_Deg[i] = ((float)rand()/RAND_MAX)*2.0f*PI;
		else
			h_In_Deg[i] = static_target;
	}
	cutilSafeCall( hipMemcpy(d_In_Deg, h_In_Deg, Tr*sizeof(float)*ind_trials, hipMemcpyHostToDevice) );


	//allocate data for theta
	float* theta = (float*)malloc(ind_trials*sizeof(float));
	float* rew = (float*)malloc(ind_trials*sizeof(float));
	int* decision_offset = (int*)malloc(ind_trials*sizeof(int));
	int* target_offset = (int*)malloc(ind_trials*sizeof(int));

	for (int n=0; n<Tr; n++)
	{


		//set initial trial specific data
		cutilSafeCall( hipMemset(d_Y, 0, T*N*sizeof(float)*ind_trials) );
		resetMembranePotential<<<NT_grid, NT_threads>>>(d_u, d_u_out);
		
		//calculate place cell distance and activations and output
		poissonNeuronSimulation<<<m_grid, m_threads>>>(n, Tr, d_X, d_Input, d_epsp, d_In_Deg, d_randSeeds, mn);
		cutilCheckMsg("Error in kernel\n");

		//for T trails (must be performed serially as lateral activation use data from t-1)
		for (int t=1; t<T; t++)
		{
			//calculate the place cell activations
			placeCellSpikePropagation<<<mN_grid, mN_threads>>>(d_W, d_Input+(t*M), d_input_pot);
			cutilCheckMsg("Error in kernel\n");

			//check if we are using dynamic simulation for lateral connections (if so calculate them)
			if (dynamics == DYN_SYS){
				//calculate the action cell lateral interactions
				actionCellLateralSpikePropagation<<<nN_grid, nN_threads>>>(d_Wr, d_Y+((t-1)*N), d_lateral_pot);
				cutilCheckMsg("Error in kernel\n");
			}

			//calculate the action cell spikes
			integrateAndFireNeuronSimulation<dynamics><<<n_grid, n_threads>>>(t, d_u, d_u_out, d_input_pot, d_lateral_pot, d_YProb, d_Y, d_randSeeds, mn);
			cutilCheckMsg("Error in kernel\n");

		
			//swap output for input
			float* d_u_temp;
			d_u_temp = d_u;
			d_u = d_u_out;
			d_u_out = d_u_temp;

		}
		

		//transpose Y
		transpose<N, T><<<NT_matrix_grid, NT_matrix_threads>>>(d_Y, d_Yt, NT_grid_width);
		cutilCheckMsg("Error in kernel\n");

		//calculate average angle of Y (for each N) across all ind trials N*ind_trials total parallel reductions
		reduceMultipleArrays<float>(T, d_Yt, d_Y_sum, N*ind_trials);
		cutilCheckMsg("Error in kernel\n");

		//calculate output angle components
		calculatePopulationVector<<<n_grid, n_threads>>>(d_Y_sum, d_out_x, d_out_y);
		cutilCheckMsg("Error in kernel\n");

		//sum output components
		reduceMultipleArrays<float>(N, d_out_x, d_out_x_sum, ind_trials);
		cutilCheckMsg("Error in kernel\n");
		reduceMultipleArrays<float>(N, d_out_y, d_out_y_sum, ind_trials);
		cutilCheckMsg("Error in kernel\n");


		//calculate angle by reading back sum totals to the CPU for each ind trial
		for (int i=0; i<ind_trials; i++)
		{
			int configuration_offset = pow2mod(i, ind_configs);
			float h_out_x_sum = 0;
			float h_out_y_sum = 0;
			int d_out_sum_offset = N*i;
			int d_In_Deg_offset = Tr*i;
			cutilSafeCall( hipMemcpy( &h_out_x_sum, d_out_x_sum+d_out_sum_offset, sizeof(float), hipMemcpyDeviceToHost) );
			cutilSafeCall( hipMemcpy( &h_out_y_sum, d_out_y_sum+d_out_sum_offset, sizeof(float), hipMemcpyDeviceToHost) );

			//calculate theta
			theta[i] = py_modf(atan2(h_out_x_sum, h_out_y_sum), 2.0f*PI);

			//calculate reward
			rew[i] = reward(abs(theta[i]-h_In_Deg[n+d_In_Deg_offset]), configuration_offset);

			//update total reward
			total_reward[i] += rew[i];
			total_error[i] += (fabs(h_In_Deg[n+d_In_Deg_offset] - theta[i]));

			//caluclate the descision and target offsets
			if (profiling == GRAPHING){
				decision_offset[i] = (int)floorf(((theta[i]+PI)*N)/(2.0f*PI));
				target_offset[i] = (int)floorf(((h_In_Deg[n+d_In_Deg_offset]+PI)*N)/(2.0f*PI));
			}
		}

		//copy rewards to device
		CUDA_SAFE_CALL( hipMemcpy( d_rew, rew, ind_trials*sizeof(float), hipMemcpyHostToDevice));
		
		//copy descision and target offsets to the device
		if (profiling == GRAPHING){
			CUDA_SAFE_CALL( hipMemcpy( d_decision_offset, decision_offset, ind_trials*sizeof(int), hipMemcpyHostToDevice));
			CUDA_SAFE_CALL( hipMemcpy( d_target_offset, target_offset, ind_trials*sizeof(int), hipMemcpyHostToDevice));
		}
		
		/* only perform the following if learning or calculating the gradiant graph*/
		if ((learning == UPDATE_LEARNING_WEIGHTS)||(profiling == GRAPHING))
		{
			//transpose rho
			transpose<N,T><<<NT_matrix_grid, NT_matrix_threads>>>(d_YProb, d_YProbt, NT_grid_width);
			cutilCheckMsg("Error in kernel\n");

			//calculate gradiant
			calculateGradiant<<<MN_matrix_grid, MN_matrix_threads>>>(MN_grid_width, d_Yt, d_YProbt, d_epsp, d_Grad);
			cutilCheckMsg("Error in kernel\n");

			if (profiling == GRAPHING)
			{
				//update the gradiant and deltaW sum totals
				updateGradiantAnalysis<<<MN_grid, MN_threads>>>(d_decision_offset, d_target_offset, d_Grad, d_rew, d_sumGrad, d_sumGrad_out, d_sumDeltaW, d_sumDeltaW_out); 
				cutilCheckMsg("Error in kernel\n");
				//swap sumGrad input and output
				float* d_sumGrad_temp;
				d_sumGrad_temp = d_sumGrad;
				d_sumGrad = d_sumGrad_out;
				d_sumGrad_out = d_sumGrad_temp;
				//swap deltaW input and output
				float* d_sumDeltaW_temp;
				d_sumDeltaW_temp = d_sumDeltaW;
				d_sumDeltaW = d_sumDeltaW_out;
				d_sumDeltaW_out = d_sumDeltaW_temp;

			}
			//no learning if we are calculating the gradiant graph
			else if (learning == UPDATE_LEARNING_WEIGHTS)
			{
				//update learning weights
				updateLearningWeights<<<MN_grid, MN_threads>>>(d_W, d_rew, d_Grad, d_W_out);
				cutilCheckMsg("Error in kernel\n");

				if (APPLY_NOISE)
				{
					//apply noise (this will also swap input and output)
					applyNoise<<<m_grid, m_threads>>>(d_W_out, d_W, d_randSeeds, mn);
					cutilCheckMsg("Error in kernel\n");
				}
				else{
					//swap the input and output pointer
					float* d_W_temp;	//used to swap input and output
					d_W_temp = d_W;
					d_W = d_W_out;
					d_W_out = d_W_temp;
				}
			}
		}
	}

	//cleanup theta
	free(theta);
	free(rew);
	free(decision_offset);
	free(target_offset);
}



void graphAnalysisDataToHost()
{
	printf("Copying graph analysis data from device to host\n");
	cutilSafeCall( hipMemcpy(h_sumGrad, d_sumGrad, N*M*sizeof(float)*ind_trials, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(h_sumDeltaW, d_sumDeltaW, N*M*sizeof(float)*ind_trials, hipMemcpyDeviceToHost) );
}

void weightsToHost(){
	printf("Copying weight data from device to host\n");
	cutilSafeCall( hipMemcpy(h_W, d_W, N*M*sizeof(float)*ind_trials, hipMemcpyDeviceToHost) );
}


/** learn_curve
 * Runs the simulation
 */
template<PROFILING profiling, DYNAMICS dynamics> 
void learn_curve() 
{

	//init model
	copyParametersToDevice();
	initLearnStepData();
	calculateGridBlockSizes();
	if ((profiling == SIMULATION_EXTENDED_ANALYSIS)||(profiling == GRAPHING)){
		initGraphAnalysis();
		printf("Analysis mode....\n");
	}

	//start a timer
    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

	//allocate arrays for reward data
	float* total_analysis_rewards = (float*) malloc(ind_trials*no_intval*sizeof(float));
	float* total_analysis_errors = (float*) malloc(ind_trials*no_intval*sizeof(float));
	float* total_reward = (float*) malloc(ind_trials*sizeof(float));
	float* total_error = (float*) malloc(ind_trials*sizeof(float));

	//calculate the number of learn trials (scale no trials for no dyn)
	int learn_trials = 0;
	if (dynamics == DYN_SYS)
		learn_trials = learn_trials_dyn;
	else //NO_DYN_SYS
		learn_trials = learn_trials_no_dyn;

	int simulation_analysis_trials;
	if (profiling == SIMULATION)
		simulation_analysis_trials = analysis_trials;
	else //SIMULATION_EXTENDED_ANALYSIS
		simulation_analysis_trials = gradiant_analsyis_trials;

	
	//PROFILE
	if(profiling == PROFILE_ONLY)
	{
		if (dynamics==DYN_SYS)
			printf("Running dyn sys profiling:\n 1 step\n single trial\n %i ind configs\n %i ind trials per config\n", ind_configs, trials_per_config);
		if (dynamics==NO_DYN_SYS)
			printf("Running no dyn sys profiling:\n 1 step\n single trial\n %i ind configs\n %i ind trials per config\n", ind_configs, trials_per_config);

		learn_step<UPDATE_LEARNING_WEIGHTS, dynamics, PROFILE_ONLY>(1, total_reward, total_error);
	}
	//CREATE GRADIANT GRAPH: no learning
	else if(profiling == GRAPHING)
	{
		if (dynamics==DYN_SYS)
			printf("Running dyn sys graph analysis:\n 1 step\n %i trials\n %i ind configs\n %i ind trials per config\n", gradiant_analsyis_trials, ind_configs, trials_per_config);
		if (dynamics==NO_DYN_SYS)
			printf("Running no dyn sys graph analysis:\n 1 step\n %i trials\n %i ind configs\n %i ind trials per config\n", gradiant_analsyis_trials, ind_configs, trials_per_config);

		//reset values
		resetGraphAnalysis();

		//perform simulation
		learn_step<ANAYLSYS_ONLY, dynamics, GRAPHING>(gradiant_analsyis_trials, total_reward, total_error);

		//copy graph analysis dat to host
		graphAnalysisDataToHost();

		//output graphs
		printf("Saving graph analysis data...\n");
		saveGraphAnalysisData<dynamics>(h_sumGrad, h_sumDeltaW);
	}
	//SIMULATION
	else	
	{
		if (dynamics==DYN_SYS)
			printf("Running dyn sys simulation:\n %i steps\n %i analysis trials\n %i learning trials\n %i ind configs\n %i ind trials per config\n", no_intval, simulation_analysis_trials, learn_trials, ind_configs, trials_per_config);
		if (dynamics==NO_DYN_SYS)
			printf("Running no dyn sys simulation:\n %i steps\n %i analysis trials\n %i learning trials\n %i ind configs\n %i ind trials per config\n", no_intval, simulation_analysis_trials, learn_trials, ind_configs, trials_per_config);
		
		for (int m=0; m<no_intval; m++)
		{
			printf("Starting Step %i of %i\n", (m+1), no_intval);
			float* no_intval_reward = &total_analysis_rewards[m*ind_trials];
			float* no_intval_error = &total_analysis_errors[m*ind_trials];

			//Perform analsyis
			if (profiling == SIMULATION)
			{
				printf("Stage 1: Performing analysis...\n");
				//perform simulation analysis
				learn_step<ANAYLSYS_ONLY, dynamics, SIMULATION>(analysis_trials, no_intval_reward, no_intval_error);
			}
			else //SIMULATION_EXTENDED_ANALYSIS
			{
				printf("Performing extended analysis...\n");
				
				//reset values
				resetGraphAnalysis();
				
				//perform extended simulation analysis
				learn_step<ANAYLSYS_ONLY, dynamics, GRAPHING>(gradiant_analsyis_trials, no_intval_reward, no_intval_error);

				//copy graph analysis data to host and produce the graph data for the current step
				graphAnalysisDataToHost();
				printf("Saving graph analysis data for step %i...\n", (m+1));
				saveGraphAnalysisData<dynamics>(h_sumGrad, h_sumDeltaW, (m+1));
			}

			//Print analsyis
			for(int j=0; j<ind_configs; j++)
			{
				for(int i=0; i<trials_per_config; i++)
				{
					printf("Step %i, Config %i, Ind trial no: %i: Av Reward is %f, Av Error %f\n", (m+1), (j+1), (i+1), no_intval_reward[j+(i*ind_configs)]/(float)simulation_analysis_trials ,no_intval_error[j+(i*ind_configs)]/(float)simulation_analysis_trials);
				}
			}

			//perform learning
			printf("Performing learn step...\n", (m+1), no_intval);
			learn_step<UPDATE_LEARNING_WEIGHTS, dynamics, SIMULATION>(learn_trials, total_reward, total_error);

			//for extended analysis print weights after each step
			if((profiling==SIMULATION_EXTENDED_ANALYSIS)&&(printWeightPlot))
			{
				weightsToHost();
				printf("Saving weight data...\n");
				saveWeightData<dynamics>(h_W, (m+1));
			}
		}
	}
	
	//stop the timer
    cutilCheckError( cutStopTimer( timer));
	printf("Simulation complete\n");
    printf("Processing time: %f (seconds)\n\n", cutGetTimerValue( timer)/1000.0f);
    cutilCheckError( cutDeleteTimer( timer));

	//output data to files ifn ot profiling
	if ((profiling == SIMULATION)||(profiling == SIMULATION_EXTENDED_ANALYSIS)){
		printf("Saving learn curve data...\n");
		saveLearnCurveData<dynamics>(total_analysis_rewards);
		saveErrorCurveData<dynamics>(total_analysis_errors);
	}

	//for simulation mode print weights once at end of simulation step
	if((profiling==SIMULATION)&&(printWeightPlot))
	{
		weightsToHost();
		printf("Saving weight data...\n");
		saveWeightData<dynamics>(h_W);
	}



	//free array for reward data
	free(total_reward);
	free(total_analysis_rewards);
	free(total_analysis_errors);

	//cleanup
	cleanupLearnStep();
	if ((profiling == SIMULATION_EXTENDED_ANALYSIS)||(profiling == GRAPHING)){
		cleanupGraphAnalysis();
	}
}


////////////////////////////////////////////////////////////////////////////////
// Program main

int
main( int argc, char** argv) 
{
	PROFILING profile;
	bool dyn_sys;
	bool no_dyn_sys;

	//init the device using command-line specified CUDA device, otherwise use device with highest Gflops/s
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

	//profile
	profile = SIMULATION;
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "profile") )
		profile = PROFILE_ONLY;
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "graph_analysis") )
		profile = GRAPHING;
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "extended_analysis") )
		profile = SIMULATION_EXTENDED_ANALYSIS;
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "print_weight_plot") )
		printWeightPlot = true;

	//check for invalid use of pwint weight plot
	if ((profile == GRAPHING)&&(printWeightPlot)){
		printf("Cannot use print_weight_plot argument with graph_analysis.\n");
		hipDeviceReset();
		cutilExit(argc, argv);
		exit(0);
	}

	
	//dyn sys (default perform both dynamics and no dynamics sequentially
	dyn_sys = true;
	no_dyn_sys = true;
	//dynamic system only
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "dyn_sys") )
		no_dyn_sys = false;
	else if( cutCheckCmdLineFlag(argc, (const char**)argv, "no_dyn_sys") )
		dyn_sys = false;

	//perform dynamic system simulation
	if (dyn_sys)
	{
		if (profile == PROFILE_ONLY)
			learn_curve<PROFILE_ONLY, DYN_SYS>();
		else if (profile == GRAPHING)
			learn_curve<GRAPHING, DYN_SYS>();
		else if (profile == SIMULATION_EXTENDED_ANALYSIS)
			learn_curve<SIMULATION_EXTENDED_ANALYSIS, DYN_SYS>();
		//SIMULTATION
		else	
			learn_curve<SIMULATION, DYN_SYS>();
	}

	//perform non dydnamic system simulation
	if(no_dyn_sys)
	{
		if (profile == PROFILE_ONLY)
			learn_curve<PROFILE_ONLY, NO_DYN_SYS>();
		else if (profile == GRAPHING)
			learn_curve<GRAPHING, NO_DYN_SYS>();
		else if (profile == SIMULATION_EXTENDED_ANALYSIS)
			learn_curve<SIMULATION_EXTENDED_ANALYSIS, NO_DYN_SYS>();
		//SIMULTATION
		else
			learn_curve<SIMULATION, NO_DYN_SYS>();
	}

	//if simulating
	if ((profile == SIMULATION)||(profile ==SIMULATION_EXTENDED_ANALYSIS))
	{
		printf("Creating learn curve graphs...\n");
		createLearnCurveGraph(dyn_sys, no_dyn_sys);
		createErrorCurveGraph(dyn_sys, no_dyn_sys);

		if (profile == SIMULATION_EXTENDED_ANALYSIS)
		{
			for (int m=0; m<no_intval; m++)
			{
				printf("Creating analysis graphs for step %i...\n", (m+1));
				createAnalysisGraphs(dyn_sys, no_dyn_sys, (m+1));

				//create a weight graph for each step
				if (printWeightPlot)
				{
					printf("Creating 3d weights plot for step %i...\n", (m+1));
					createWeightGraphs(dyn_sys, no_dyn_sys, (m+1));
				}

			}
		}
		else //SIMULATION
		{
			//create a single weight plot graph
			if (printWeightPlot)
			{
				printf("Creating 3d weights plot...\n");
				createWeightGraphs(dyn_sys, no_dyn_sys);
			}
		}
	}
	//if performing graph analysis
	else if (profile == GRAPHING)
	{
		printf("Creating analysis graphs...\n");
		createAnalysisGraphs(dyn_sys, no_dyn_sys);
	}


	

	hipDeviceReset();

	//this will pause the window if we are not profiling
	if(profile != PROFILE_ONLY)
		cutilExit(argc, argv);
}
