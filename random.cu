/*
  Copyright (c) 2011 Paul Richmond, University of Sheffield , UK; 
  all rights reserved unless otherwise stated.

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  In addition to the regulations of the GNU General Public License,
  publications and communications based in parts on this program or on
  parts of this program are required to cite the article 
  "Democratic population decisions result in robust policy-gradient 
  learning: a parametric study with GPU simulations" by Paul Richmond, 
  Lars Buesing, Michele Giugliano and Eleni Vasilaki, PLoS ONE Neuroscience, 
  Under Review.. 

  This program is distributed in the hope that it will be useful, but
  WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
  General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston,
  MA 02111-1307 USA
*/

/*
This CUDA source file provides the necessary host functions for initialising 
an algorithm implementing random number generation on the GPU. The 
algorithm is based upon that described in the article "Harvesting 
graphics power for MD simulations" by J.A. van Meel, A. Arnold, 
D. Frenkel, S. F. Portegies Zwart and R. G. Belleman, Molecular Simulation, 
Vol. 34, p. 259 (2007) distributed under the GNU GPL v2.
*/

#include <cutil_inline.h>
#include "random.h"


/* Host code init function */
void initCUDARand48(unsigned int max_rand, rand48seeds* h_seeds, rand48seeds* d_seeds, magic_numbers &mn)
{
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	srand ( (unsigned int) time(NULL) );
	int seed = rand();
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	
	for (unsigned int i = 0; i < max_rand; ++i) {
		C += A*c;
		A *= a;
	}
	
	//magic numbers
	mn.x = A & 0xFFFFFFLL;
	mn.y = (A >> 24) & 0xFFFFFFLL;
	mn.z = C & 0xFFFFFFLL;
	mn.w = (C >> 24) & 0xFFFFFFLL;

	//prepare MAX_RAND numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i=0; i<max_rand; i++)
	{
		x = a*x + c;
		h_seeds[i].x = x & 0xFFFFFFLL;
		h_seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}

	//copy seeds to device
	CUDA_SAFE_CALL( hipMemcpy( d_seeds, h_seeds, max_rand*sizeof(rand48seeds), hipMemcpyHostToDevice));

}



