/*
  Copyright (c) 2011 Paul Richmond, University of Sheffield , UK; 
  all rights reserved unless otherwise stated.

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  In addition to the regulations of the GNU General Public License,
  publications and communications based in parts on this program or on
  parts of this program are required to cite the article 
  "Democratic population decisions result in robust policy-gradient 
  learning: a parametric study with GPU simulations" by Paul Richmond, 
  Lars Buesing, Michele Giugliano and Eleni Vasilaki, PLoS ONE Neuroscience, 
  Under Review.. 

  This program is distributed in the hope that it will be useful, but
  WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
  General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston,
  MA 02111-1307 USA
*/

/*
  "This software contains source code provided by NVIDIA Corporation." 
  This CUDA source file defines the host functions used for parallel 
  reduction and is based on the CUDA SDK Parallel reduction example 
  provided with the CUDA Computing SDK. It has been modified to allow a 
  large number of simultaneous (and independent) parallel reductions. i.e.
  multiple reductions.

  The reduction kernel is used for both the spikeTrainRedeuction and outputComponentReduction 
  steps of the simulation.
*/


#ifndef _REDUCTION_H_
#define _REDUCTION_H_

#include "reduction.cuh"

bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}



void getReductionBlocksAndThreads(int n, int &blocks, int &threads)
{
	threads = (n < MAX_REDUCTION_THREADS*2) ? nextPow2((n + 1)/ 2) : MAX_REDUCTION_THREADS;
    blocks = (n + (threads * 2 - 1)) / (threads * 2);
	blocks = MIN(MAX_REDUCTION_BLOCKS, blocks);
}




template <class T>
void 
reduce(int size, int threads, int blocks, T *d_idata, T *d_odata, int multiple = 1, int total_size = 0)
{
	dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, multiple, 1);
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

	//if total size is not default then use mutiple reductions kernel
	if (multiple > 1)
	{
		if (isPow2(size))
		{
			switch (threads)
			{
			case 512:
				reduce6_multiple<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 256:
				reduce6_multiple<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 128:
				reduce6_multiple<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 64:
				reduce6_multiple<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 32:
				reduce6_multiple<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 16:
				reduce6_multiple<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  8:
				reduce6_multiple<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  4:
				reduce6_multiple<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  2:
				reduce6_multiple<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  1:
				reduce6_multiple<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			}
		}
		else
		{
			switch (threads)
			{
			case 512:
				reduce6_multiple<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 256:
				reduce6_multiple<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 128:
				reduce6_multiple<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 64:
				reduce6_multiple<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 32:
				reduce6_multiple<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case 16:
				reduce6_multiple<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  8:
				reduce6_multiple<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  4:
				reduce6_multiple<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  2:
				reduce6_multiple<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			case  1:
				reduce6_multiple<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, total_size); break;
			}
		}
	}
	//only require a single reduction
	else
	{
		if (isPow2(size))
		{
			switch (threads)
			{
			case 512:
				reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 256:
				reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 128:
				reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 64:
				reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 32:
				reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 16:
				reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  8:
				reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  4:
				reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  2:
				reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  1:
				reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			}
		}
		else
		{
			switch (threads)
			{
			case 512:
				reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 256:
				reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 128:
				reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 64:
				reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 32:
				reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 16:
				reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  8:
				reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  4:
				reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  2:
				reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  1:
				reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			}
		}
	}
}


template <class T>
T reduceArray(int n, T* d_idata, T* d_odata)
{
    T gpu_result;

	int nThreads = 0;
	int nBlocks = 0;
	getReductionBlocksAndThreads(n, nBlocks, nThreads);

    //single reduction
	reduce<T>(n, nThreads, nBlocks, d_idata, d_odata);


    // check if kernel execution generated an error
    cutilCheckMsg("Kernel execution failed");

    
    // sum partial block sums on GPU
    int s=nBlocks;
    while(s > 1) 
    {
        int threads = 0, blocks = 0;
        getReductionBlocksAndThreads(s, blocks, threads);
        
		//single reduction
		reduce<T>(s, threads, blocks, d_odata, d_odata);

        s = (s + (threads*2-1)) / (threads*2);

    }
        
	cutilSafeCallNoSync( hipMemcpy( &gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost) );

    return gpu_result;
}

template <class T>
void reduceMultipleArrays(int n, T* d_idata, T* d_odata, int multiple)
{

	int nThreads = 0;
	int nBlocks = 0;
	getReductionBlocksAndThreads(n, nBlocks, nThreads);

    //mutiple parallel reductions
	reduce<T>(n, nThreads, nBlocks, d_idata, d_odata, multiple, n);

    // check if kernel execution generated an error
    cutilCheckMsg("Kernel execution failed");

    
    // sum partial block sums on GPU
    int s=nBlocks;
    while(s > 1) 
    {
        int threads = 0, blocks = 0;
        getReductionBlocksAndThreads(s, blocks, threads);
        
		//mutiple reductions
		reduce<T>(s, threads, blocks, d_odata, d_odata, multiple, n);
        
        s = (s + (threads*2-1)) / (threads*2);

    }
        
}



template int 
reduceArray<int>(int n, int* d_idata, int* d_odata);

template float 
reduceArray<float>(int n, float* d_idata, float* d_odata);

template double 
reduceArray<double>(int n, double* d_idata, double* d_odata);

template void 
reduceMultipleArrays<int>(int n, int* d_idata, int* d_odata, int multiple);

template void 
reduceMultipleArrays<float>(int n, float* d_idata, float* d_odata, int multiple);

template void 
reduceMultipleArrays<double>(int n, double* d_idata, double* d_odata, int multiple);

#endif //_REDUCTION_H_