#include "hip/hip_runtime.h"
/*
  Copyright (c) 2011 Paul Richmond, University of Sheffield , UK; 
  all rights reserved unless otherwise stated.

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  In addition to the regulations of the GNU General Public License,
  publications and communications based in parts on this program or on
  parts of this program are required to cite the article 
  "Democratic population decisions result in robust policy-gradient 
  learning: a parametric study with GPU simulations" by Paul Richmond, 
  Lars Buesing, Michele Giugliano and Eleni Vasilaki, PLoS ONE Neuroscience, 
  Under Review.. 

  This program is distributed in the hope that it will be useful, but
  WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
  General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston,
  MA 02111-1307 USA
*/
/*
This CUDA source file contains the function definitions used for creating data 
plots and generating GPU plot scripts.
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <windows.h>
#include <time.h>
#include <float.h>
#include <cutil_inline.h>

#include "output.cuh"

template<DYNAMICS dynamics>
void printConfigurationToFile(FILE* stream, int config_no)
{
	time_t rawtime;
	struct tm * timeinfo;
			
	//output configuration info
	time ( &rawtime );
	timeinfo = localtime ( &rawtime );
	fprintf(stream, "# Output from GPU model: %s", asctime(timeinfo));
	fprintf(stream, "# Configuration number (%i)\n", config_no);
	if (dynamics == DYN_SYS)
		fprintf(stream, "# Simulation using Dynamical Lateral Connections (%i)\n", config_no);
	else //NO_DYN_SYS
		fprintf(stream, "# Simulation NOT using Dynamical Lateral Connections (%i)\n", config_no);
	if (moving_target)
		fprintf(stream, "# Random moving target between trials\n");
	else
		fprintf(stream, "# Static moving target (%f) between trials\n", static_target);
	if (REFACTORY)
		fprintf(stream, "# Refactory period for spiking neurons\n");
	else
		fprintf(stream, "# No refactory period for spiking neurons\n");

	fprintf(stream, "# \n");
	fprintf(stream, "# Global Values ************************\n");
	fprintf(stream, "# N = %i \n", N);
	fprintf(stream, "# M = %i \n", M);
	fprintf(stream, "# T = %i \n", T);

	fprintf(stream, "# Learning Step Values ************************\n");
	fprintf(stream, "# no_intval = %i \n", no_intval);
	fprintf(stream, "# analysis_trials = %i \n", analysis_trials);
	fprintf(stream, "# gradiant_analysis_trials = %i \n", gradiant_analsyis_trials);
	fprintf(stream, "# learn_trials_dyn_sys = %i \n", learn_trials_dyn);
	fprintf(stream, "# learn_trials_no_dyn_sys = %i \n", learn_trials_no_dyn);
	fprintf(stream, "# trials_per_config = %i \n", trials_per_config);

	fprintf(stream, "# Simulation Parameters ****************\n");
	fprintf(stream, "# sig_p = %f \n", sig_p[config_no]);
	fprintf(stream, "# rsc = %f \n", rsc[config_no]);
	fprintf(stream, "# w_E = %f \n", w_E[config_no]);
	if (reward_func == GAUSSIAN){
		fprintf(stream, "# GAUSSIAN REWARD FUNCTION\n");
		fprintf(stream, "# sigma_R = %f \n", sigma_R[config_no]);
	}else{	//BOX
		fprintf(stream, "# BOX REWARD FUNCTION\n");
		fprintf(stream, "# sigma_R_box = %f \n", sigma_R_box[config_no]);
	}
	fprintf(stream, "# beta = %f \n", h_beta[config_no]);
	fprintf(stream, "# xsc = %f \n", h_xsc[config_no]);
	fprintf(stream, "# w0 = %f \n", h_w0[config_no]);
	fprintf(stream, "# u0 = %f \n", h_u0[config_no]);
	fprintf(stream, "# eta = %f \n", h_eta[config_no]);
	fprintf(stream, "# baseline = %f \n", h_baseline[config_no]);
	fprintf(stream, "# \n\n\n");

	fprintf(stream, "# Spiking Neuron Parameters ****************\n");
	fprintf(stream, "# dt = %f \n", dt);
	fprintf(stream, "# tau = %f \n", h_tau[config_no]);
	fprintf(stream, "# lambda = %f \n", h_lambda[config_no]);
	fprintf(stream, "# urest = %f \n", h_urest[config_no]);
	fprintf(stream, "# threshold = %f \n", h_threshold[config_no]);
	fprintf(stream, "# du = %f \n", h_du[config_no]);
	fprintf(stream, "# rho0 = %f \n", h_rho0[config_no]);
	fprintf(stream, "# ref = %f \n", h_ref[config_no]);

}

template<DYNAMICS dynamics>
void saveLearnCurveData(float* total_analysis_rewards)
{
	for (int f = 0; f< ind_configs; f++)
	{
		char filename[128];
		FILE* ar;
		if (dynamics == DYN_SYS)
			sprintf(filename, "%s/%s_average_reward(dyn_sys).dat", config_output_dir, config_output_prefix[f]);
		else //NO_DYN_SYS
			sprintf(filename, "%s/%s_average_reward(no_dyn_sys).dat", config_output_dir, config_output_prefix[f]);
		ar = fopen(filename, "w");
		if (ar == NULL){
			printf("Error: Can't open output file %s!\n", filename);
			exit(0);
		}

		printConfigurationToFile<dynamics>(ar, f);

		fprintf(ar, "# Average Reward Data ******************\n");
		fprintf(ar, "# Step\tMEAN_REWARD\tSTD DEVIATION\n");
		
		float average_total = 0.0f;
		float sd_total = 0.0f;
		//calculate averages
		for (int m=0; m<no_intval; m++)
		{
			float average_reward = 0;
			float std_dev = 0;

			//For each step of each config calculate the average accross the ind trials
			for(int i=0; i<trials_per_config; i++)
			{
				float* no_intval_reward = &total_analysis_rewards[m*ind_trials];
				float reward = no_intval_reward[f+(i*ind_configs)]/(float)analysis_trials;
				average_reward += reward;
			}
			average_reward /= (float)trials_per_config;

			//For each step of each config calculate the SD accross the ind trials
			for(int i=0; i<trials_per_config; i++)
			{
				float* no_intval_reward = &total_analysis_rewards[m*ind_trials];
				float reward = no_intval_reward[f+(i*ind_configs)]/(float)analysis_trials;
				std_dev += (average_reward - reward)*(average_reward - reward);
			}
			std_dev /= (float)trials_per_config;
			std_dev = sqrtf(std_dev);


			//print to data file with error bars
			fprintf(ar, "%i\t%f\t%f\n", m, average_reward, std_dev);

			average_total += average_reward;
			sd_total += std_dev;
		}

		average_total /= no_intval;
		sd_total /= no_intval;
		fprintf(ar, "# average over steps\n");
		fprintf(ar, "# %f\t%f\n", average_total, sd_total);

		//close the file
		fclose(ar);
	}
}

template<DYNAMICS dynamics>
void saveErrorCurveData(float* total_reward_errors)
{
	for (int f = 0; f< ind_configs; f++)
	{
		char filename[128];
		FILE* ar;
		if (dynamics == DYN_SYS)
			sprintf(filename, "%s/%s_reward_error(dyn_sys).dat", config_output_dir, config_output_prefix[f]);
		else //NO_DYN_SYS
			sprintf(filename, "%s/%s_reward_error(no_dyn_sys).dat", config_output_dir, config_output_prefix[f]);
		ar = fopen(filename, "w");
		if (ar == NULL){
			printf("Error: Can't open output file %s!\n", filename);
			exit(0);
		}

		printConfigurationToFile<dynamics>(ar, f);

		fprintf(ar, "# Reward Error Data ******************\n");
		fprintf(ar, "# Step\tMEAN_REWARD\tSTD DEVIATION\n");
		
		float error_total = 0.0f;
		float sd_total = 0.0f;
		//calculate averages
		for (int m=0; m<no_intval; m++)
		{
			float average_error = 0;
			float std_dev = 0;

			//For each step of each config calculate the average accross the ind trials
			for(int i=0; i<trials_per_config; i++)
			{
				float* no_intval_reward = &total_reward_errors[m*ind_trials];
				float error = no_intval_reward[f+(i*ind_configs)]/(float)analysis_trials;
				average_error += error;
			}
			average_error /= (float)trials_per_config;

			//For each step of each config calculate the SD accross the ind trials
			for(int i=0; i<trials_per_config; i++)
			{
				float* no_intval_reward = &total_reward_errors[m*ind_trials];
				float error = no_intval_reward[f+(i*ind_configs)]/(float)analysis_trials;
				std_dev += (average_error - error)*(average_error - error);
			}
			std_dev /= (float)trials_per_config;
			std_dev = sqrtf(std_dev);


			//print to data file with error bars
			fprintf(ar, "%i\t%f\t%f\n", m, average_error, std_dev);

			error_total += average_error;
			sd_total += std_dev;
		}

		error_total /= no_intval;
		sd_total /= no_intval;
		fprintf(ar, "# average over steps\n");
		fprintf(ar, "# %f\t%f\n", error_total, sd_total);

		//close the file
		fclose(ar);
	}
}

void createLearnCurveGraph(bool dyn_sys, bool no_dyn_sys)
{
	for (int f = 0; f< ind_configs; f++)
	{
		char filename[128];
		char dyn_sys_filename[128];
		char no_dyn_sys_filename[128];
		time_t rawtime;
		struct tm * timeinfo;
		FILE* ar;

		//file names
		sprintf(filename, "%s/%s_reward_error.plt", config_output_dir, config_output_prefix[f]);
		ar = fopen(filename, "w");
		if (ar == NULL){
			printf("Error: Can't open output file %s!\n", filename);
			exit(0);
		}

		//files used for graph input
		sprintf(dyn_sys_filename, "%s_reward_error(dyn_sys).dat", config_output_prefix[f]);
		sprintf(no_dyn_sys_filename, "%s_reward_error(no_dyn_sys).dat", config_output_prefix[f]);

		//output file header information
		time ( &rawtime );
		timeinfo = localtime ( &rawtime );
		fprintf(ar, "# Output from GPU model: %s", asctime(timeinfo));
		fprintf(ar, "# GNUPlot Script");
		fprintf(ar, "set title 'Average Reward Error Plot for %s'\n", config_output_prefix[f]);
		fprintf(ar, "set xlabel 'learning steps'\n");
		fprintf(ar, "set ylabel 'reward error'\n");

		//create plot
		if (dyn_sys && no_dyn_sys){
			fprintf(ar, "plot '%s' with lines, '%s'  with yerrorbars title 'SD accross ind trials', '%s' with lines, '%s' with yerrorbars title 'SD accross ind trials'",	dyn_sys_filename, dyn_sys_filename, no_dyn_sys_filename, no_dyn_sys_filename);
		}
		else
		{
			if(dyn_sys)
				fprintf(ar, "plot '%s' with lines, '%s' with yerrorbars title 'SD accross ind trials",	dyn_sys_filename, dyn_sys_filename, no_dyn_sys_filename, no_dyn_sys_filename);
			if(no_dyn_sys)
				fprintf(ar, "plot '%s' with lines, '%s' with yerrorbars title 'SD accross ind trials", no_dyn_sys_filename, no_dyn_sys_filename);
		}
		fclose(ar);
	}

}


void createErrorCurveGraph(bool dyn_sys, bool no_dyn_sys)
{
	for (int f = 0; f< ind_configs; f++)
	{
		char filename[128];
		char dyn_sys_filename[128];
		char no_dyn_sys_filename[128];
		time_t rawtime;
		struct tm * timeinfo;
		FILE* ar;

		//file names
		sprintf(filename, "%s/%s_average_reward.plt", config_output_dir, config_output_prefix[f]);
		ar = fopen(filename, "w");
		if (ar == NULL){
			printf("Error: Can't open output file %s!\n", filename);
			exit(0);
		}

		//files used for graph input
		sprintf(dyn_sys_filename, "%s_average_reward(dyn_sys).dat", config_output_prefix[f]);
		sprintf(no_dyn_sys_filename, "%s_average_reward(no_dyn_sys).dat", config_output_prefix[f]);

		//output file header information
		time ( &rawtime );
		timeinfo = localtime ( &rawtime );
		fprintf(ar, "# Output from GPU model: %s", asctime(timeinfo));
		fprintf(ar, "# GNUPlot Script");
		fprintf(ar, "set title 'Average Reward Plot for %s'\n", config_output_prefix[f]);
		fprintf(ar, "set xlabel 'learning steps'\n");
		fprintf(ar, "set ylabel 'average reward'\n");

		//create plot
		if (dyn_sys && no_dyn_sys){
			fprintf(ar, "plot '%s' with lines, '%s'  with yerrorbars title 'SD accross ind trials', '%s' with lines, '%s' with yerrorbars title 'SD accross ind trials'",	dyn_sys_filename, dyn_sys_filename, no_dyn_sys_filename, no_dyn_sys_filename);
		}
		else
		{
			if(dyn_sys)
				fprintf(ar, "plot '%s' with lines, '%s' with yerrorbars title 'SD accross ind trials",	dyn_sys_filename, dyn_sys_filename, no_dyn_sys_filename, no_dyn_sys_filename);
			if(no_dyn_sys)
				fprintf(ar, "plot '%s' with lines, '%s' with yerrorbars title 'SD accross ind trials", no_dyn_sys_filename, no_dyn_sys_filename);
		}
		fclose(ar);
	}

}

template<DYNAMICS dynamics>
void saveGraphAnalysisData(float* h_sumGrad, float* h_sumDeltaW, int step)
{
	char step_str[128] = "";
	if (step > 0)
	{
		sprintf(step_str, "_step_%i", step);
	}
	for (int f = 0; f< ind_configs; f++)
	{
		//create and open files for writing mean gradiant and mean deltaW
		char mG_filename[128];
		char dW_filename[128];
		FILE* mG;
		FILE* dW;
		if (dynamics == DYN_SYS){
			sprintf(mG_filename, "%s/%s_gradiant(dyn_sys)%s.dat", config_output_dir, config_output_prefix[f], step_str);
			sprintf(dW_filename, "%s/%s_deltaW(dyn_sys)%s.dat", config_output_dir, config_output_prefix[f], step_str);
		}
		else{ //NO_DYN_SYS
			sprintf(mG_filename, "%s/%s_gradiant(no_dyn_sys)%s.dat", config_output_dir, config_output_prefix[f], step_str);
			sprintf(dW_filename, "%s/%s_deltaW(no_dyn_sys)%s.dat", config_output_dir, config_output_prefix[f], step_str);
		}
		mG = fopen(mG_filename, "w");
		if (mG == NULL){
			printf("Error: Can't open output file %s!\n", mG_filename);
			exit(0);
		}
		dW = fopen(dW_filename, "w");
		if (dW == NULL){
			printf("Error: Can't open output file %s!\n", dW_filename);
			exit(0);
		}

		//output file header information
		printConfigurationToFile<dynamics>(mG, f);
		printConfigurationToFile<dynamics>(dW, f);
		
		//loop through action cells
		for (int n=0; n<N; n++)
		{
			float avr_grad = 0.0f;	
			float avr_deltaW = 0.0f;	
			//average for each neuron connection (xM) and over the indipendant trials
			for(int i=0; i<trials_per_config; i++)
			{
				//offset for each indipendant trial
				int trial_offset = (f+(i*ind_configs))*M*N;
				for (int m=0; m<M; m++)
				{
					avr_grad += h_sumGrad[(n*M) + m + trial_offset];
					avr_deltaW += h_sumDeltaW[(n*M) + m + trial_offset];
				}
			}
			//average
			avr_grad /= M*trials_per_config;
			avr_deltaW /= M*trials_per_config;
			
			//print to data files
			fprintf(mG, "%i\t%f\n", n, avr_grad);
			fprintf(dW, "%i\t%f\n", n, avr_deltaW);
		}

		//close the files
		fclose(mG);
		fclose(dW);
	}
}

void createAnalysisGraphs(bool dyn_sys, bool no_dyn_sys, int step)
{
	char step_str[128] = "";
	if (step > 0)
	{
		sprintf(step_str, "_step_%i", step);
	}

	for (int f = 0; f< ind_configs; f++)
	{
		char mG_filename[128];
		char dW_filename[128];
		char dyn_sys_mG_filename[128];
		char no_dyn_sys_mG_filename[128];
		char dyn_sys_dW_filename[128];
		char no_dyn_sys_dW_filename[128];
		time_t rawtime;
		struct tm * timeinfo;
		FILE* mG;
		FILE* dW;

		//file names
		sprintf(mG_filename, "%s/%s_gradiant%s.plt", config_output_dir, config_output_prefix[f], step_str);
		sprintf(dW_filename, "%s/%s_deltaW%s.plt", config_output_dir, config_output_prefix[f], step_str);
		mG = fopen(mG_filename, "w");
		if (mG == NULL){
			printf("Error: Can't open output file %s!\n", mG_filename);
			exit(0);
		}
		dW = fopen(dW_filename, "w");
		if (dW == NULL){
			printf("Error: Can't open output file %s!\n", dW_filename);
			exit(0);
		}

		//files used for graph input
		sprintf(dyn_sys_mG_filename, "%s_gradiant(dyn_sys)%s.dat", config_output_prefix[f], step_str);
		sprintf(no_dyn_sys_mG_filename, "%s_gradiant(no_dyn_sys)%s.dat", config_output_prefix[f], step_str);
		sprintf(dyn_sys_dW_filename, "%s_deltaW(dyn_sys)%s.dat", config_output_prefix[f], step_str);
		sprintf(no_dyn_sys_dW_filename, "%s_deltaW(no_dyn_sys)%s.dat", config_output_prefix[f], step_str);

		//get time info
		time ( &rawtime );
		timeinfo = localtime ( &rawtime );

		//output file header infor for gradiant mean
		fprintf(mG, "# Output from GPU model: %s", asctime(timeinfo));
		fprintf(mG, "# GNUPlot Script");
		fprintf(mG, "# %i gradiant anyalysis trials", gradiant_analsyis_trials);
		fprintf(mG, "# %i indipendant trials per configuration", trials_per_config);
		fprintf(mG, "# Step %i of %i", step, no_intval);
		fprintf(mG, "set title 'Action Cells Mean Gradiant (before learning)%s'\n", config_output_prefix[f]);
		fprintf(mG, "set xlabel 'Action Cell Index (aligned by decision)'\n");
		fprintf(mG, "set ylabel 'mean gradiant'\n");

		//output file header infor for delta W
		fprintf(dW, "# Output from GPU model: %s", asctime(timeinfo));
		fprintf(dW, "# GNUPlot Script");
		fprintf(dW, "# %i gradiant anyalysis trials", gradiant_analsyis_trials);
		fprintf(dW, "# %i indipendant trials per configuration", trials_per_config);
		fprintf(dW, "set title 'Action Cells Delta W (before learning)%s'\n", config_output_prefix[f]);
		fprintf(dW, "set xlabel 'Action Cell Index (aligned by target angle and offset 180*)'\n");
		fprintf(dW, "set ylabel 'delta W'\n");

		//create plot
		if (dyn_sys && no_dyn_sys){
			fprintf(mG, "plot '%s' with lines, '%s' with lines",	dyn_sys_mG_filename, no_dyn_sys_mG_filename);
			fprintf(dW, "plot '%s' with lines, '%s' with lines",	dyn_sys_dW_filename, no_dyn_sys_dW_filename);
		}
		else
		{
			if(dyn_sys){
				fprintf(mG, "plot '%s' with lines", dyn_sys_mG_filename);
				fprintf(dW, "plot '%s' with lines", dyn_sys_dW_filename);
			}
			if(no_dyn_sys){
				fprintf(mG, "plot '%s' with lines", no_dyn_sys_mG_filename);
				fprintf(dW, "plot '%s' with lines", no_dyn_sys_dW_filename);
			}
		}

		//close files
		fclose(mG);
		fclose(dW);
	}

}


template<DYNAMICS dynamics>
void saveWeightData(float* h_W, int step)
{
char step_str[128] = "";
	if (step > 0)
	{
		sprintf(step_str, "_step_%i", step);
	}
	for (int f = 0; f< ind_configs; f++)
	{
		//create and open files for writing mean gradiant and mean deltaW
		char w_filename[128];
		FILE* w;
		if (dynamics == DYN_SYS){
			sprintf(w_filename, "%s/%s_W(dyn_sys)%s.dat", config_output_dir, config_output_prefix[f], step_str);
		}
		else{ //NO_DYN_SYS
			sprintf(w_filename, "%s/%s_W(no_dyn_sys)%s.dat", config_output_dir, config_output_prefix[f], step_str);
		}
		w = fopen(w_filename, "w");
		if (w == NULL){
			printf("Error: Can't open output file %s!\n", w_filename);
			exit(0);
		}

		//output file header information
		printConfigurationToFile<dynamics>(w, f);

		
		//loop through action cells
		for (int n=0; n<N; n++)
		{
			for (int m=0; m<M; m++)
			{
				float avr_w = 0;
				for(int i=0; i<trials_per_config; i++)
				{
					//offset for each indipendant trial
					int trial_offset = (f+(i*ind_configs))*M*N;

					avr_w += h_W[(n*M) + m + trial_offset];
				}
				avr_w /= trials_per_config;
				fprintf(w, "%i\t%i\t%f\n", n, m, avr_w);
			}
			fprintf(w, "\n");
		}

		//close the files
		fclose(w);
	}

}

void createWeightGraphs(bool dyn_sys, bool no_dyn_sys, int step)
{
	char step_str[128] = "";
	if (step > 0)
	{
		sprintf(step_str, "_step_%i", step);
	}

	for (int f = 0; f< ind_configs; f++)
	{
		char dyn_sys_w_plot_filename[128];
		char dyn_sys_w_filename[128];
		char no_dyn_sys_w_plot_filename[128];
		char no_dyn_sys_w_filename[128];
		time_t rawtime;
		struct tm * timeinfo;
		FILE* w_dyn_sys;
		FILE* w_no_dyn_sys;

		//get time info
		time ( &rawtime );
		timeinfo = localtime ( &rawtime );

		if(dyn_sys){

			//file names
			sprintf(dyn_sys_w_plot_filename, "%s/%s_W(dyn_sys)%s.plt", config_output_dir, config_output_prefix[f], step_str);
			w_dyn_sys = fopen(dyn_sys_w_plot_filename, "w");
			if (w_dyn_sys == NULL){
				printf("Error: Can't open output file %s!\n", dyn_sys_w_plot_filename);
				exit(0);
			}

			//files used for graph input
			sprintf(dyn_sys_w_filename, "%s_W(dyn_sys)%s.dat", config_output_prefix[f], step_str);

			//output file header infor for gradiant mean
			fprintf(w_dyn_sys, "# Output from GPU model: %s", asctime(timeinfo));
			fprintf(w_dyn_sys, "# GNUPlot Script");
			fprintf(w_dyn_sys, "# %i gradiant anyalysis trials", gradiant_analsyis_trials);
			fprintf(w_dyn_sys, "# %i indipendant trials per configuration", trials_per_config);
			fprintf(w_dyn_sys, "# Step %i of %i", step, no_intval);
			fprintf(w_dyn_sys, "set title 'Action Cell Weight: Step %i %s'\n", step, config_output_prefix[f]);
			fprintf(w_dyn_sys, "set xlabel 'N'\n");
			fprintf(w_dyn_sys, "set ylabel 'M'\n");	

			fprintf(w_dyn_sys, "plot '%s' with image", dyn_sys_w_filename);

			fclose(w_dyn_sys);

		}

		
		if(no_dyn_sys){
			sprintf(no_dyn_sys_w_plot_filename, "%s/%s_W(no_dyn_sys)%s.plt", config_output_dir, config_output_prefix[f], step_str);
			w_no_dyn_sys = fopen(no_dyn_sys_w_plot_filename, "w");
			if (w_no_dyn_sys == NULL){
				printf("Error: Can't open output file %s!\n", no_dyn_sys_w_plot_filename);
				exit(0);
			}

			sprintf(no_dyn_sys_w_filename, "%s_W(no_dyn_sys)%s.dat", config_output_prefix[f], step_str);

			fprintf(w_no_dyn_sys, "# Output from GPU model: %s", asctime(timeinfo));
			fprintf(w_no_dyn_sys, "# GNUPlot Script");
			fprintf(w_no_dyn_sys, "# %i gradiant anyalysis trials", gradiant_analsyis_trials);
			fprintf(w_no_dyn_sys, "# %i indipendant trials per configuration", trials_per_config);
			fprintf(w_no_dyn_sys, "# Step %i of %i", step, no_intval);
			fprintf(w_no_dyn_sys, "set title 'Action Cell Weight: Step %i %s'\n", step, config_output_prefix[f]);
			fprintf(w_no_dyn_sys, "set xlabel 'N'\n");
			fprintf(w_no_dyn_sys, "set ylabel 'M'\n");

			fprintf(w_no_dyn_sys, "plot '%s' with image", no_dyn_sys_w_filename);

			fclose(w_no_dyn_sys);
		}
	}

}

//template prototypes
template void saveLearnCurveData<NO_DYN_SYS>(float* total_analysis_rewards);
template void saveLearnCurveData<DYN_SYS>(float* total_analysis_rewards);

template void saveErrorCurveData<NO_DYN_SYS>(float* total_reward_errors);
template void saveErrorCurveData<DYN_SYS>(float* total_reward_errors);

template void saveGraphAnalysisData<NO_DYN_SYS>(float* h_sumGrad, float* h_sumDeltaW, int step);
template void saveGraphAnalysisData<DYN_SYS>(float* h_sumGrad, float* h_sumDeltaW, int step);

template void saveWeightData<NO_DYN_SYS>(float* h_W, int step);
template void saveWeightData<DYN_SYS>(float* h_W, int step);

